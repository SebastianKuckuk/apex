#include "dmvp-util.h"
#include "../../cuda-util.h"


#include <hipblas.h>


#define CUBLAS_CHECK(status) \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS Error: " << status << std::endl; \
        exit(1); \
    }


template <typename tpe>
inline int realMain(int argc, char *argv[]) {
    char *tpeName;
    size_t nx, nItWarmUp, nIt;
    parseCLA_1d(argc, argv, tpeName, nx, nItWarmUp, nIt);

    double *mat, *src, *dest;
    checkCudaError(hipHostMalloc((void **) &mat, sizeof(double) * nx * nx));
    checkCudaError(hipHostMalloc((void **) &src, sizeof(double) * nx));
    checkCudaError(hipHostMalloc((void **) &dest, sizeof(double) * nx));

    // init
    initDMVP(mat, src, nx);

    double *d_mat, *d_src, *d_dest;
    checkCudaError(hipMalloc((void **) &d_mat, sizeof(double) * nx * nx));
    checkCudaError(hipMalloc((void **) &d_src, sizeof(double) * nx));
    checkCudaError(hipMalloc((void **) &d_dest, sizeof(double) * nx));

    checkCudaError(hipMemcpy(d_mat, mat, sizeof(double) * nx * nx, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_src, src, sizeof(double) * nx, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_dest, dest, sizeof(double) * nx, hipMemcpyHostToDevice));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        const auto alpha = 1.;
        const auto beta = 0.;
        CUBLAS_CHECK(hipblasDgemv(handle, HIPBLAS_OP_N, nx, nx, &alpha, d_mat, nx, d_src, 1, &beta, d_dest, 1));
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        const auto alpha = 1.;
        const auto beta = 0.;
        CUBLAS_CHECK(hipblasDgemv(handle, HIPBLAS_OP_N, nx, nx, &alpha, d_mat, nx, d_src, 1, &beta, d_dest, 1));
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    // mem: matrix, dest, src; flops: 1 FMA per matrix entry
    printStatsDMVP<tpe>(end - start, nIt, nx, tpeName, nx * sizeof(tpe) + sizeof(tpe) + sizeof(tpe), 2 * nx);

    checkCudaError(hipMemcpy(src, d_src, sizeof(double) * nx, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(dest, d_dest, sizeof(double) * nx, hipMemcpyDeviceToHost));

    // check solution
    checkSolutionDMVP(src, nx, nIt + nItWarmUp);

    checkCudaError(hipFree(d_mat));
    checkCudaError(hipFree(d_src));
    checkCudaError(hipFree(d_dest));

    checkCudaError(hipHostFree(mat));
    checkCudaError(hipHostFree(src));
    checkCudaError(hipHostFree(dest));

    return 0;
}


int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Missing type specification" << std::endl;
        return -1;
    }

    std::string tpeName(argv[1]);

    if ("float" == tpeName)
        return realMain<float>(argc, argv);
    if ("double" == tpeName)
        return realMain<double>(argc, argv);

    std::cout << "Invalid type specification (" << argv[1] << "); supported types are" << std::endl;
    std::cout << "  float, double" << std::endl;
    return -1;
}
