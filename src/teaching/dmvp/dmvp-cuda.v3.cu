#include "hip/hip_runtime.h"
#include "dmvp-util.h"
#include "../../cuda-util.h"


template <typename tpe>
__global__ void dmvp(size_t nx, const tpe *const __restrict__ mat, const tpe *const __restrict__ src, tpe *__restrict__ dest) {
    auto rStart = blockIdx.x * blockDim.x + threadIdx.x;
    auto rStride = gridDim.x * blockDim.x;
    auto cStart = blockIdx.y * blockDim.y + threadIdx.y;
    auto cStride = gridDim.y * blockDim.y;

    for (size_t r = rStart; r < nx; r += rStride) {
        auto acc = 0.;
        for (size_t c = cStart; c < nx; c += cStride)
            acc += mat[r * nx + c] * src[c];
        atomicAdd(&dest[r], acc);
    }
}


template <typename tpe>
inline int realMain(int argc, char *argv[]) {
    char *tpeName;
    size_t nx, nItWarmUp, nIt;
    parseCLA_1d(argc, argv, tpeName, nx, nItWarmUp, nIt);

    double *mat, *src, *dest;
    checkCudaError(hipHostMalloc((void **) &mat, sizeof(double) * nx * nx));
    checkCudaError(hipHostMalloc((void **) &src, sizeof(double) * nx));
    checkCudaError(hipHostMalloc((void **) &dest, sizeof(double) * nx));

    // init
    initDMVP(mat, src, nx);

    double *d_mat, *d_src, *d_dest;
    checkCudaError(hipMalloc((void **) &d_mat, sizeof(double) * nx * nx));
    checkCudaError(hipMalloc((void **) &d_src, sizeof(double) * nx));
    checkCudaError(hipMalloc((void **) &d_dest, sizeof(double) * nx));

    checkCudaError(hipMemcpy(d_mat, mat, sizeof(double) * nx * nx, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_src, src, sizeof(double) * nx, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpy(d_dest, dest, sizeof(double) * nx, hipMemcpyHostToDevice));

    dim3 blockSize(16, 16);
    dim3 numBlocks(128, 128);

    // warm-up
    for (size_t i = 0; i < nItWarmUp; ++i) {
        hipMemset(d_dest, 0, sizeof(double) * nx);
        dmvp<<<numBlocks, blockSize>>>(nx, d_mat, d_src, d_dest);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    // measurement
    auto start = std::chrono::steady_clock::now();

    for (size_t i = 0; i < nIt; ++i) {
        hipMemset(d_dest, 0, sizeof(double) * nx);
        dmvp<<<numBlocks, blockSize>>>(nx, d_mat, d_src, d_dest);
        std::swap(d_src, d_dest);
    }
    checkCudaError(hipDeviceSynchronize(), true);

    auto end = std::chrono::steady_clock::now();

    // mem: matrix, dest, src; flops: 1 FMA per matrix entry
    printStatsDMVP<tpe>(end - start, nIt, nx, tpeName, nx * sizeof(tpe) + sizeof(tpe) + sizeof(tpe), 2 * nx);

    checkCudaError(hipMemcpy(src, d_src, sizeof(double) * nx, hipMemcpyDeviceToHost));
    checkCudaError(hipMemcpy(dest, d_dest, sizeof(double) * nx, hipMemcpyDeviceToHost));

    // check solution
    checkSolutionDMVP(src, nx, nIt + nItWarmUp);

    checkCudaError(hipFree(d_mat));
    checkCudaError(hipFree(d_src));
    checkCudaError(hipFree(d_dest));

    checkCudaError(hipHostFree(mat));
    checkCudaError(hipHostFree(src));
    checkCudaError(hipHostFree(dest));

    return 0;
}


int main(int argc, char *argv[]) {
    if (argc < 2) {
        std::cout << "Missing type specification" << std::endl;
        return -1;
    }

    std::string tpeName(argv[1]);

    if ("float" == tpeName)
        return realMain<float>(argc, argv);
    if ("double" == tpeName)
        return realMain<double>(argc, argv);

    std::cout << "Invalid type specification (" << argv[1] << "); supported types are" << std::endl;
    std::cout << "  float, double" << std::endl;
    return -1;
}
